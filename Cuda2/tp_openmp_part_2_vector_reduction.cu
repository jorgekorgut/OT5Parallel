/*
//@HEADER
// ************************************************************************
//
//                        Kokkos v. 2.0
//              Copyright (2014) Sandia Corporation
//
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions Contact  H. Carter Edwards (hcedwar@sandia.gov)
//
// ************************************************************************
//@HEADER
*/


#include <hip/hip_runtime.h>
#include <limits>

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <iostream>
#include <cmath>
#include <float.h>
#include <assert.h>
#include <execinfo.h>

// CUDA global constants
// Allocate x,y,A
#define VariableType double
#define VariableTypePlus double

__device__ VariableType *d_y;
__device__ VariableType *d_A;
__device__ VariableType *d_x;

__global__ void calculate(VariableType *d_sum, VariableType *d_A, VariableType *d_x, VariableType *d_y, int N, int M);
__device__ void calculateSlice(VariableType *d_A, VariableType *d_x, VariableType *d_y, int a, int b, int N, int M);
void checkCudaError(int id);

void checkSizes(int &N, int &M, int &S, int &nrepeat);

int main(int argc, char *argv[])
{
  int N = -1;      // number of rows 2^12
  int M = -1;      // number of columns 2^10
  int S = -1;      // total size 2^22
  int nrepeat = 1; // number of repeats of the test

  // Read command line arguments.
  for (int i = 0; i < argc; i++)
  {
    if ((strcmp(argv[i], "-N") == 0) || (strcmp(argv[i], "-Rows") == 0))
    {
      N = pow(2, atoi(argv[++i]));
      fprintf(stderr, "  User N is %d\n", N);
    }
    else if ((strcmp(argv[i], "-M") == 0) || (strcmp(argv[i], "-Columns") == 0))
    {
      M = pow(2, atof(argv[++i]));
      fprintf(stderr, "  User M is %d\n", M);
    }
    else if ((strcmp(argv[i], "-S") == 0) || (strcmp(argv[i], "-Size") == 0))
    {
      S = pow(2, atof(argv[++i]));
      fprintf(stderr, "  User S is %d\n", S);
    }
    else if (strcmp(argv[i], "-nrepeat") == 0)
    {
      nrepeat = atoi(argv[++i]);
    }
    else if ((strcmp(argv[i], "-h") == 0) || (strcmp(argv[i], "-help") == 0))
    {
      fprintf(stderr, "  y^T*A*x Options:\n");
      fprintf(stderr, "  -Rows (-N) <int>:      exponent num, determines number of rows 2^num (default: 2^12 = 4096)\n");
      fprintf(stderr, "  -Columns (-M) <int>:   exponent num, determines number of columns 2^num (default: 2^10 = 1024)\n");
      fprintf(stderr, "  -Size (-S) <int>:      exponent num, determines total matrix size 2^num (default: 2^22 = 4096*1024 )\n");
      fprintf(stderr, "  -nrepeat <int>:        number of repetitions (default: 100)\n");
      fprintf(stderr, "  -help (-h):            print this message\n\n");
      exit(1);
    }
  }

  // Check sizes.
  checkSizes(N, M, S, nrepeat);

  // Initialize y vector to 1.
  VariableType *y = new VariableType[N];
  for (int i = 0; i < N; i++)
  {
    y[i] = 1;
  }

  hipMalloc(&d_y, sizeof(VariableType) * N);
  checkCudaError(1);

  // Initialize x column vector to 1 .

  VariableType *x = new VariableType[M];
  for (int i = 0; i < M; i++)
  {
    x[i] = 1;
  }
  hipMalloc(&d_x, sizeof(VariableType) * M);
  hipMemcpy(d_x, x, sizeof(VariableType) * M, hipMemcpyHostToDevice);
  checkCudaError(2);

  //  Initialize A matrix, you can use a 1D index if you want a flat structure (i.e. a 1D array) e.g. j*M+i is the same than [j][i]
  VariableType *A = new VariableType[N * M];

  for (int i = 0; i < N * M; i++)
  {
    A[i] = 1;
  }
  hipMalloc(&d_A, sizeof(VariableType) * N * M);
  hipMemcpy(d_A, A, sizeof(VariableType) * N * M, hipMemcpyHostToDevice);
  checkCudaError(3);

  // Cuda parameters
  int num_blocks = N;
  int threadPerBlock = 256;

  int resultByteSize = sizeof(VariableType);
  VariableType *h_sum = (VariableType *)malloc(resultByteSize);
  VariableType *d_sum;
  hipMalloc(&d_sum, resultByteSize);
  checkCudaError(4);

  // Timer products.
  struct timeval begin, end;

  gettimeofday(&begin, NULL);

  for (int repeat = 0; repeat < nrepeat; repeat++)
  {

    *h_sum = 0;
    hipMemcpy(d_sum, h_sum, resultByteSize, hipMemcpyHostToDevice);

    calculate<<<num_blocks, threadPerBlock>>>(d_sum, d_A, d_x, d_y, N, M);

    hipMemcpy(h_sum, d_sum, resultByteSize, hipMemcpyDeviceToHost);

    VariableType result = *h_sum;

    checkCudaError(5 + repeat);

    // Output result.
    if (repeat == (nrepeat - 1))
    {
      fprintf(stderr, "  Computed result for %d x %d is %lf\n", N, M, result);
    }

    const double solution = (double)N * (double)M;

    if (result != solution)
    {
      fprintf(stderr, "  Error: result( %lf ) != solution( %lf )\n", result, solution);
    }
  }

  gettimeofday(&end, NULL);

  // Calculate time.
  // double time = timer.seconds();
  double time = 1.0 * (end.tv_sec - begin.tv_sec) +
                1.0e-6 * (end.tv_usec - begin.tv_usec);

  // Calculate bandwidth.
  // Each matrix A row (each of length M) is read once.
  // The x vector (of length M) is read N times.
  // The y vector (of length N) is read once.
  // double Gbytes = 1.0e-9 * double( sizeof(double) * ( 2 * M * N + N ) );
  double Gbytes = 1.0e-9 * double(sizeof(double) * (M + M * N + N));

  // Print results (problem size, time and bandwidth in GB/s).
  fprintf(stderr, "  N( %d ) M( %d ) nrepeat ( %d ) problem( %g MB ) time( %g s ) bandwidth( %g GB/s )\n",
          N, M, nrepeat, Gbytes * 1000, time, Gbytes * nrepeat / time);

  std::cout << time;

  std::free(A);
  std::free(y);
  std::free(x);

  return 0;
}

void checkSizes(int &N, int &M, int &S, int &nrepeat)
{
  // If S is undefined and N or M is undefined, set S to 2^22 or the bigger of N and M.
  if (S == -1 && (N == -1 || M == -1))
  {
    S = pow(2, 22);
    if (S < N)
      S = N;
    if (S < M)
      S = M;
  }

  // If S is undefined and both N and M are defined, set S = N * M.
  if (S == -1)
    S = N * M;

  // If both N and M are undefined, fix row length to the smaller of S and 2^10 = 1024.
  if (N == -1 && M == -1)
  {
    if (S > 1024)
    {
      M = 1024;
    }
    else
    {
      M = S;
    }
  }

  // If only M is undefined, set it.
  if (M == -1)
    M = S / N;

  // If N is undefined, set it.
  if (N == -1)
    N = S / M;

  fprintf(stderr, "  Total size S = %d N = %d M = %d\n", S, N, M);

  // Check sizes.
  if ((S < 0) || (N < 0) || (M < 0) || (nrepeat < 0))
  {
    fprintf(stderr, "  Sizes must be greater than 0.\n");
    exit(1);
  }

  if ((N * M) != S)
  {
    fprintf(stderr, "  N * M != S\n");
    exit(1);
  }
}

__global__ void calculate(VariableType *d_sum, VariableType *d_A, VariableType *d_x, VariableType *d_y, int N, int M)
{
  int unique_id = blockDim.x * blockIdx.x + threadIdx.x;

  if (unique_id < N)
  {
    // int rowId = blockIdx.x;
    int elementId = M * unique_id;

    calculateSlice(d_A, d_x, d_y, elementId, elementId + M, N, M);

    atomicAdd(d_sum, d_y[unique_id]);
  }
}

__device__ void calculateSlice(VariableType *d_A, VariableType *d_x, VariableType *d_y, int a, int b, int N, int M)
{
  int startRow = a / M;
  int startColumn = a % M;

  int endRow = b / M + 1;
  int endColumn = b % M;

  if (endColumn == 0)
  {
    endRow--;
    endColumn = M;
  }

  // printf("a %d, b %d\n", a, b);
  //printf("startRow: %d, startColumn: %d, endRow: %d, endColumn: %d, d_A: %f\n", startRow, startColumn, endRow, endColumn, d_A[0]);

  for (int i = startRow; i < endRow; i++)
  {

    VariableTypePlus partialSum = 0;
    for (int j = startColumn; j < endColumn; j++)
    {
      // printf("i: %d, j: %d, d_A: %f, d_x: %f\n", i, j, d_A[i * M + j], d_x[j]);
      partialSum += d_A[i * M + j] * d_x[j];
      // printf("partialSum: %f\n", partialSum);
    }
    // printf("partialSum: %f\n", partialSum);

    d_y[i] = partialSum;
    // atomicAdd(d_y + i, partialSum);
  }
}

void checkCudaError(int id)
{
  hipError_t error = hipGetLastError();
  if (error)
  {
    const char *errorName = hipGetErrorName(error);
    const char *errorDescription = hipGetErrorString(error);

    printf("%s : %s ", errorName, errorDescription);
    printf("id: %d\n", id);
    assert(1 == 2);
  }
}
