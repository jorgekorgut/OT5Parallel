#include "hip/hip_runtime.h"
/*

This program will numerically compute the integral of

                  4/(1+x*x) 
				  
from 0 to 1.  The value of this integral is pi -- which 
is great since it gives us an easy way to check the answer.

History: Written by Tim Mattson, 11/1999.
         Modified/extended by Jonathan Rouzaud-Cornabas, 10/2022
*/

#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <algorithm>

__global__
void calculatePi(double* pi, double step, int num_steps, int threadSize, int num_threads);

__device__ 
double calculatePartialPi(int a, int b, double step, int num_steps);

static long num_steps = 1e8;

int main (int argc, char** argv)
{
    
      // Read command line arguments.
      for ( int i = 0; i < argc; i++ ) {
        if ( ( strcmp( argv[ i ], "-N" ) == 0 ) || ( strcmp( argv[ i ], "-num_steps" ) == 0 ) ) {
            num_steps = atol( argv[ ++i ] );
            printf( "  User num_steps is %ld\n", num_steps );
        } else if ( ( strcmp( argv[ i ], "-h" ) == 0 ) || ( strcmp( argv[ i ], "-help" ) == 0 ) ) {
            printf( "  Pi Options:\n" );
            printf( "  -num_steps (-N) <int>:      Number of steps to compute Pi (by default 100000000)\n" );
            printf( "  -help (-h):            print this message\n\n" );
            exit( 1 );
        }
      }
	  
    double step = 1.0/(double) num_steps;

    // Timer products.
    struct timeval begin, end;

    gettimeofday( &begin, NULL );
    
    int size =sizeof(double);
    double * h_sum = (double*)malloc(size);
    *h_sum = 0;
    double * d_sum;
    hipMalloc(&d_sum, size);
    hipMemcpy(d_sum, h_sum, size, hipMemcpyHostToDevice);

    int num_blocks = 1024;
    int threadSize = num_steps/num_blocks + 1;
    int num_threads = 4;

    calculatePi<<<num_blocks,num_threads>>>(d_sum, step, num_steps, threadSize, num_threads);

    //hipDeviceSynchronize();
    hipMemcpy(h_sum, d_sum, size, hipMemcpyDeviceToHost);

    double pi = step * (*h_sum);
	  
    gettimeofday( &end, NULL );

    // Calculate time.
    double time = 1.0 * ( end.tv_sec - begin.tv_sec ) +
                1.0e-6 * ( end.tv_usec - begin.tv_usec );
                
    printf("\n pi with %ld steps is %lf in %lf seconds\n ",num_steps,pi,time);
}

extern __shared__ double local_sum[num_steps/2+1];

__global__
void calculatePi(double* sum, double step, int num_steps, int threadSize, int num_threads)
{
  int i = (blockIdx.x*num_threads + threadIdx.x) * threadSize + 1;

  double partialSum = calculatePartialPi(i,i+threadSize, step, num_steps);
  
  atomicAdd(sum,partialSum);
}

__device__  
double calculatePartialPi(int a, int b, double step, int num_steps)
{
  
  double x, sum = 0;
  int i;
  for(i=a; i<b && i < num_steps ; i++){
      x = (i-0.5)*step;
      
		  sum = sum + 4.0/(1.0+x*x);
  }
  
  return sum;
}